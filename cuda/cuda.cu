#include "hip/hip_runtime.h"

#include <math.h>
#include <data.h>
#include <velocity_verlet.h>
// #include <../src/output.c>
// #include <../src/input.c>
// #include <../src/utilities.c>
// #include <../src/verlet1.c>
// #include <../src/verlet2.c>

/* compute forces */

__global__ void focres(double * d_rx,double *d_ry,double * d_rz, double * d_vx,double * d_vy,double * d_vz,double * d_fx,\
  double * d_fy,double * d_fz,double epot, double  epsilon,double sigma,int natoms, double box, double rcut){

    double ffac,r2,r6,r12,sigma2;
    double rx,ry,rz;
    // int i,j,n;

    sigma2 = sigma*sigma;

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int idy = threadIdx.y + blockIdx.y*blockDim.y;

    epot = 0.0;
     d_fx[idx] = 0.0;
     d_fy[idx] = 0.0;
     d_fz[idx] = 0.0;


        // if (i==j) continue;
        if (idx > idy){
            while ((d_rx[idx] - d_rx[idy]) >  0.5*box) rx -= box; while ((d_rx[idx] - d_rx[idy]) <  0.5*box) rx += box;
            while ((d_ry[idx] - d_ry[idy]) >  0.5*box) ry -= box; while ((d_ry[idx] - d_ry[idy]) <  0.5*box) ry += box;
            while ((d_rz[idx] - d_rz[idy]) >  0.5*box) rz -= box; while ((d_rz[idx] - d_rz[idy]) <  0.5*box) rz += box;

            r2 = rx*rx + ry*ry + rz*rz;

            if (r2 < rcut*rcut ) {
                r2 = (sigma2)/r2;
                r6 = r2*r2*r2;
                r12 = r6*r6;

                ffac = -4.0*epsilon*(-12.0*r12+6*r6)*r2/sigma2;
                epot += 4.0*epsilon*(r12-r6);

                d_fx[idx] += rx*ffac; d_fx[idy] -= rx*ffac;
                d_fy[idx] += ry*ffac; d_fy[idy] -= ry*ffac;
                d_fz[idx] += rz*ffac; d_fz[idy] -= rz*ffac;
            }
        }
    }
